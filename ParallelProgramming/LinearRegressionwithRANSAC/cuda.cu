
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <vector>
#include <cmath>
#include <random>
#include <algorithm>
#include <fstream>
#include <chrono>
using namespace std;
#define N 100000000
int n;
double points[N];
double true_slope, true_intercept;
double best_slope, best_intercept;
const int K =  1000*2; // num of iteration
const double T = 1e-4; // threshold
const int B = 32; // blolck size (max num of thread per block)

__global__ void calculateInliers(double *d_points,  bool*d_inliers, double threshold, double slope, double intercept, int n, int iter, size_t pitch){
    int point_idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(point_idx >= n) return;
    double expected_y = slope * d_points[2*point_idx] + intercept;
    double error = fabs(d_points[2*point_idx+1] - expected_y);
    d_inliers[pitch * iter + point_idx] = (error < threshold);
}

__global__ void countInliers(int *d_inliers_cnt,  bool*d_inliers, int n ,int max_iter,size_t pitch){
    const int iter_idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(iter_idx >= max_iter) return;
    int inliers_cnt = 0;
    for(int i=0;i<n;i++){
        if(d_inliers[pitch*iter_idx + i]){
            inliers_cnt++;
        }
    }
    d_inliers_cnt[iter_idx] = inliers_cnt; 
}

__global__ void findBestInliers(int *d_inliers_cnt, int* d_best_idx, int max_iter){
    int best_idx = -1;
    int best_inliers_cnt = 0;
    int cnt = 0;
    for(int i = 0; i< max_iter; i++){
        cnt = d_inliers_cnt[i];
        if (cnt > best_inliers_cnt) {
            best_inliers_cnt = cnt;
            best_idx = i;
        }
    }
    d_best_idx[0] = best_idx;
}

// Function to perform 2D linear regression using RANSAC
void linearRegressionRANSAC(int iterations, double threshold) {
    mt19937 gen(0);
    uniform_int_distribution<int> dist(0, n - 1);

    int inliers[n];
    int best_inliers_cnt = 0;
    int best_inlier[n];
    double *d_points;
    bool *d_inliers;


    // allocate GPU memory for points arrya & inlier matrix 
    hipMalloc((void**)&d_points, 2 * n * sizeof(double));
    hipMemcpy(d_points, points, 2 * n * sizeof(double), hipMemcpyHostToDevice);

    size_t pitch = n;
	//cudaMallocPitch((void**)&d_inliers, &pitch, n * K * sizeof(bool), n);
    hipMalloc((void**)&d_inliers, n * K * sizeof(bool));

    int block_size = B;
    dim3 block_dim(block_size, 1);
    int grid_size = int( ceil(float(n)/float(B)) );
    dim3 grid_dim(grid_size, 1);
    cout << "block size: " << block_size << endl;
    cout << "grid size: " << grid_size  << endl;
    double x1, y1, x2, y2;
    int idx1, idx2;
    double slope, intercept;

    for (int i = 0; i < iterations; i++) {
        // Randomly select two points
        idx1 = dist(gen);
        idx2 = dist(gen);

        x1 = points[2*idx1];
        y1 = points[2*idx1 + 1];
        x2 = points[2*idx2];
        y2 = points[2*idx2 + 1];

        // Calculate slope and intercept
        slope = (y2 - y1) / (x2 - x1 + 1e-4);
        intercept = y1 - slope * x1;

        // check to see if each point is inlier //GPU O(n) -> O(1)
        calculateInliers<<<grid_dim, block_dim>>>(d_points, d_inliers, threshold, slope, intercept, n, i , pitch);
        hipError_t err = hipGetLastError();
        if(err!=hipSuccess){
            cout << "cuda error:" << hipGetErrorString(err) << endl;
        }        
    }

    int* d_inliers_cnt;
    int* d_best_idx;
    // malloc memory on GPU for inliers count of each round (K rounds)
    hipMalloc((void**)&d_inliers_cnt, K * sizeof(int));
    // malloc memory on GPU for max inliers index
    hipMalloc((void**)&d_best_idx, sizeof(int));
  
    auto start = chrono::steady_clock::now();
    // calculate inliers cnt for each round  //GPU
    block_size = B;
    grid_size = int( ceil(float(K)/float(B)) );
    dim3 grid_dim_2(grid_size, 1);
    dim3 block_dim_2(block_size, 1);
    countInliers<<<grid_dim_2, block_dim_2>>>(d_inliers_cnt,  d_inliers, n , K, pitch);

    // find best inliers //GPU
    block_size = 1;
    grid_size = 1;
    dim3 grid_dim_3(grid_size, 1);
    dim3 block_dim_3(block_size, 1);
    findBestInliers<<<grid_dim_3, block_dim_3>>>(d_inliers_cnt, d_best_idx, K);

    // copy best inliers indices back to host
    int* best_idx = (int*)malloc(sizeof(int));
    hipMemcpy(best_idx, d_best_idx, sizeof(int), hipMemcpyDeviceToHost);
    bool* is_inlier = (bool*)malloc(n * sizeof(bool));
    hipMemcpy(is_inlier, d_inliers + pitch*(*best_idx), n * sizeof(bool), hipMemcpyDeviceToHost);

    // CPU
    best_inliers_cnt = 0;
    for(int i=0;i<n;i++){
        if(is_inlier[i]){
            best_inlier[best_inliers_cnt++] = i;
        }
    }

    auto end = chrono::steady_clock::now();
    cout << "best inliers count: " << best_inliers_cnt << endl;
    cout << "best index: " << *best_idx << endl;
    cout << "===================================" << endl;
    cout << "Time for finding best inliers: " << chrono::duration_cast<chrono::milliseconds>(end - start).count() << " millisecond." << endl;
    cout << "===================================" << endl;
    
    hipFree(d_points);
    hipFree(d_inliers);
    hipFree(d_inliers_cnt);
    hipFree(d_best_idx);


    // Regression using best inliers
    double sum_x = 0, sum_y = 0, sum_xy = 0, sum_xx = 0;

    for (int i = 0; i < best_inliers_cnt; i++) {
        double x_val = points[2*best_inlier[i]];
        double y_val = points[2*best_inlier[i] + 1];
        sum_x += x_val;
        sum_y += y_val;
        sum_xy += x_val * y_val;
        sum_xx += x_val * x_val;
    }
    
    double mean_x = sum_x / best_inliers_cnt;
    double mean_y = sum_y / best_inliers_cnt;

    best_slope = (sum_xy - best_inliers_cnt * mean_x * mean_y) / (sum_xx - best_inliers_cnt * mean_x * mean_x);
    best_intercept = mean_y - best_slope * mean_x;
    
    return;
}

int main(int argc, char *argv[]) {
    //input
    ifstream infile(argv[1]);

    infile >> true_slope >> true_intercept;

    int i = 0;
    double x, y;
    while (infile >> x >> y){
        points[i++] = x;
        points[i++] = y;
    }
    n = (i + 1) / 2;
    cout << "Count: " << n << endl;
    auto start = chrono::steady_clock::now();
    linearRegressionRANSAC(K, T);
    auto end = chrono::steady_clock::now();

    // Display the result
    cout << "===================================" << endl;
    cout << "Estimated Slope: " << best_slope << endl;
    cout << "Estimated Intercept: " << best_intercept << endl;
    cout << "===================================" << endl;
    cout << "Ground Truth Slope: " << true_slope << endl;
    cout << "Ground Truth Intercept: " << true_intercept << endl;
    cout << "===================================" << endl;
    cout << "Time for RANSAC: " << chrono::duration_cast<chrono::milliseconds>(end - start).count() << " millisecond." << endl;
    cout << "Time for RANSAC: " << chrono::duration_cast<chrono::microseconds>(end - start).count() << " microsecond." << endl;
    return 0;
}