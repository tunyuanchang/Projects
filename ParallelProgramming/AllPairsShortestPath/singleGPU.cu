#include <stdio.h>
#include <stdlib.h>
#include <cassert>
#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

#define BLOCK_SIZE 64
#define HALF_BLOCK_SIZE BLOCK_SIZE/2

__global__ void Phase1(int *dist, int Round, int n);
__global__ void Phase2(int *dist, int Round, int n);
__global__ void Phase3(int *dist, int Round, int n);

int n, m, _n;
const int INF = 1073741823;
int *Dist = NULL;

int ceil(int a, int b){ return (a + b - 1) / b;}

void block_FW(){
    int *dist = NULL;

    const unsigned long matrixSize = n * n * sizeof(int);
    hipHostRegister(Dist, matrixSize, hipHostRegisterDefault);
    hipMalloc(&dist, matrixSize);
    hipMemcpy(dist, Dist, matrixSize, hipMemcpyHostToDevice);

    const int num_blocks = ceil(n, BLOCK_SIZE);
    dim3 block_dim(HALF_BLOCK_SIZE, HALF_BLOCK_SIZE, 1);
    dim3 grid_dim(num_blocks, num_blocks, 1);

    const int round = ceil(n, BLOCK_SIZE);
    
    for (int r = 0; r < round; ++r){
        Phase1<<<1, block_dim>>>(dist, r, n);
        Phase2<<<num_blocks, block_dim>>>(dist, r, n);
        Phase3<<<grid_dim, block_dim>>>(dist, r, n);
    }

    hipMemcpy(Dist, dist, matrixSize, hipMemcpyDeviceToHost);
    hipFree(dist);
    
    return;
}

__global__ void Phase1(int *dist, int Round, int n){
    __shared__ int shared_memory[BLOCK_SIZE][BLOCK_SIZE];
    int i = threadIdx.y;
    int j = threadIdx.x;
    int half_i = i + HALF_BLOCK_SIZE;
    int half_j = j + HALF_BLOCK_SIZE;
    int offset = BLOCK_SIZE * Round;

    shared_memory[i][j] = dist[offset * (n + 1) + i * n + j];
    shared_memory[half_i][j] = dist[offset * (n + 1) + half_i * n + j];
    shared_memory[i][half_j] = dist[offset * (n + 1) + i * n + half_j];
    shared_memory[half_i][half_j] = dist[offset * (n + 1) + half_i * n + half_j];
    __syncthreads();

    for (int k = 0; k < BLOCK_SIZE; k++){
        shared_memory[i][j] = min(shared_memory[i][j], shared_memory[i][k] + shared_memory[k][j]);
        shared_memory[half_i][j] = min(shared_memory[half_i][j], shared_memory[half_i][k] + shared_memory[k][j]);
        shared_memory[i][half_j] = min(shared_memory[i][half_j], shared_memory[i][k] + shared_memory[k][half_j]);
        shared_memory[half_i][half_j] = min(shared_memory[half_i][half_j], shared_memory[half_i][k] + shared_memory[k][half_j]);
        __syncthreads();
    }

    dist[offset * (n + 1) + i * n + j] = shared_memory[i][j];
    dist[offset * (n + 1) + half_i * n + j] = shared_memory[half_i][j];
    dist[offset * (n + 1) + i * n + half_j] = shared_memory[i][half_j];
    dist[offset * (n + 1) + half_i * n + half_j] = shared_memory[half_i][half_j];
    
    return;
}

__global__ void Phase2(int *dist, int Round, int n){
    __shared__ int pivot[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int row_blocks[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int col_blocks[BLOCK_SIZE][BLOCK_SIZE];

    int i = threadIdx.y;
    int j = threadIdx.x;
    int block_i = blockIdx.x;
    int half_i = i + HALF_BLOCK_SIZE;
    int half_j = j + HALF_BLOCK_SIZE;
    int offset = BLOCK_SIZE * Round;

    if (block_i == Round) return;

    row_blocks[i][j] = dist[(block_i * BLOCK_SIZE + i) * n + offset + j];
    row_blocks[half_i][j] = dist[(block_i * BLOCK_SIZE + half_i) * n + offset + j];
    row_blocks[i][half_j] = dist[(block_i * BLOCK_SIZE + i) * n + offset + half_j];
    row_blocks[half_i][half_j] = dist[(block_i * BLOCK_SIZE + half_i) * n + offset + half_j];

    col_blocks[i][j] = dist[(offset + i) * n + (block_i * BLOCK_SIZE + j)];
    col_blocks[half_i][j] = dist[(offset + half_i) * n + (block_i * BLOCK_SIZE + j)];
    col_blocks[i][half_j] = dist[(offset + i) * n + (block_i * BLOCK_SIZE + half_j)];
    col_blocks[half_i][half_j] = dist[(offset + half_i) * n + (block_i * BLOCK_SIZE + half_j)];

    pivot[i][j] = dist[offset * (n + 1) + i * n + j];
    pivot[half_i][j] = dist[offset * (n + 1) + half_i * n + j];
    pivot[i][half_j] = dist[offset * (n + 1) + i * n + half_j];
    pivot[half_i][half_j] = dist[offset * (n + 1) + half_i * n + half_j];
    __syncthreads();

    for (int k = 0; k < BLOCK_SIZE; k++){
        row_blocks[i][j] = min(row_blocks[i][j], row_blocks[i][k] + pivot[k][j]);
        row_blocks[half_i][j] = min(row_blocks[half_i][j], row_blocks[half_i][k] + pivot[k][j]);
        row_blocks[i][half_j] = min(row_blocks[i][half_j], row_blocks[i][k] + pivot[k][half_j]);
        row_blocks[half_i][half_j] = min(row_blocks[half_i][half_j], row_blocks[half_i][k] + pivot[k][half_j]);

        col_blocks[i][j] = min(col_blocks[i][j], pivot[i][k] + col_blocks[k][j]);
        col_blocks[half_i][j] = min(col_blocks[half_i][j], pivot[half_i][k] + col_blocks[k][j]);
        col_blocks[i][half_j] = min(col_blocks[i][half_j], pivot[i][k] + col_blocks[k][half_j]);
        col_blocks[half_i][half_j] = min(col_blocks[half_i][half_j], pivot[half_i][k] + col_blocks[k][half_j]);
    }

    dist[(block_i * BLOCK_SIZE + i) * n + offset + j] = row_blocks[i][j];
    dist[(block_i * BLOCK_SIZE + half_i) * n + offset + j] = row_blocks[half_i][j];
    dist[(block_i * BLOCK_SIZE + i) * n + offset + half_j] = row_blocks[i][half_j];
    dist[(block_i * BLOCK_SIZE + half_i) * n + offset + half_j] = row_blocks[half_i][half_j];

    dist[(offset + i)* n + (block_i * BLOCK_SIZE + j)] = col_blocks[i][j];
    dist[(offset + half_i) * n + (block_i * BLOCK_SIZE + j)] = col_blocks[half_i][j];
    dist[(offset + i) * n + (block_i * BLOCK_SIZE + half_j)] = col_blocks[i][half_j];
    dist[(offset + half_i) * n + (block_i * BLOCK_SIZE + half_j)] = col_blocks[half_i][half_j];
    
    return;
}

__global__ void Phase3(int *dist, int Round, int n){
    __shared__ int row_blocks[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int col_blocks[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int shared_memory[BLOCK_SIZE][BLOCK_SIZE];

    int i = threadIdx.y;
    int j = threadIdx.x;
    int block_i = blockIdx.y;
    int block_j = blockIdx.x;
    int half_i = i + HALF_BLOCK_SIZE;
    int half_j = j + HALF_BLOCK_SIZE;
    int offset = BLOCK_SIZE * Round;

    if (block_i == Round && block_j == Round) return;

    shared_memory[i][j] = dist[(block_i * BLOCK_SIZE + i) * n + (block_j * BLOCK_SIZE + j)];
    shared_memory[half_i][j] = dist[(block_i * BLOCK_SIZE + half_i) * n + (block_j * BLOCK_SIZE + j)];
    shared_memory[i][half_j] = dist[(block_i * BLOCK_SIZE + i) * n + (block_j * BLOCK_SIZE + half_j)];
    shared_memory[half_i][half_j] = dist[(block_i * BLOCK_SIZE + half_i) * n + (block_j * BLOCK_SIZE + half_j)];

    row_blocks[i][j] = dist[(block_i * BLOCK_SIZE + i) * n + offset + j];
    row_blocks[half_i][j] = dist[(block_i * BLOCK_SIZE + half_i) * n + offset + j];
    row_blocks[i][half_j] = dist[(block_i * BLOCK_SIZE + i) * n + offset + half_j];
    row_blocks[half_i][half_j] = dist[(block_i * BLOCK_SIZE + half_i) * n + offset + half_j];

    col_blocks[i][j] = dist[(offset + i) * n + (block_j * BLOCK_SIZE + j)];
    col_blocks[half_i][j] = dist[(offset + half_i) * n + (block_j * BLOCK_SIZE + j)];
    col_blocks[i][half_j] = dist[(offset + i) * n + (block_j * BLOCK_SIZE + half_j)];
    col_blocks[half_i][half_j] = dist[(offset + half_i) * n + (block_j * BLOCK_SIZE + half_j)];
    __syncthreads();

    for (int k = 0; k < BLOCK_SIZE; k++){
        shared_memory[i][j] = min(shared_memory[i][j], row_blocks[i][k] + col_blocks[k][j]);
        shared_memory[half_i][j] = min(shared_memory[half_i][j], row_blocks[half_i][k] + col_blocks[k][j]);
        shared_memory[i][half_j] = min(shared_memory[i][half_j], row_blocks[i][k] + col_blocks[k][half_j]);
        shared_memory[half_i][half_j] = min(shared_memory[half_i][half_j], row_blocks[half_i][k] + col_blocks[k][half_j]);
    }

    dist[(block_i * BLOCK_SIZE + i) * n + (block_j * BLOCK_SIZE + j)] = shared_memory[i][j];
    dist[(block_i * BLOCK_SIZE + half_i) * n + (block_j * BLOCK_SIZE + j)] = shared_memory[half_i][j];
    dist[(block_i * BLOCK_SIZE + i) * n + (block_j * BLOCK_SIZE + half_j)] = shared_memory[i][half_j];
    dist[(block_i * BLOCK_SIZE + half_i) * n + (block_j * BLOCK_SIZE + half_j)] = shared_memory[half_i][half_j];
    
    return;
}

int main(int argc, char *argv[]){
    // input
    FILE *infile = fopen(argv[1], "rb");
    if(infile == NULL) cout << "Fail to open file" << endl; 
    
    fread(&_n, sizeof(int), 1, infile);
    fread(&m, sizeof(int), 1, infile);
    
    n = BLOCK_SIZE * ceil(_n, BLOCK_SIZE); // padding, let n % BLOCK_SIZE == 0
    Dist = (int *)malloc(n * n * sizeof(int));
    
    for (int i = 0; i < n; ++i){
        for (int j = 0; j < n; ++j){
            if (i == j) Dist[i * n + j] = 0;
            else Dist[i * n + j] = INF;
        }
    }

    int pair[3];
    while(m--){
        fread(pair, sizeof(int), 3, infile);
        Dist[pair[0] * n + pair[1]] = pair[2];
    }
    fclose(infile);
    
    // calculate
    block_FW();

    // output
    FILE *outfile = fopen(argv[2], "w");
    for (int i = 0; i < _n; ++i){
        fwrite(&Dist[i * n], sizeof(int), _n, outfile);
    }
    fclose(outfile);

    hipHostFree(Dist);
    return 0;
}